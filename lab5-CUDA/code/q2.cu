#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>

const double alpha = 1.0;
const double beta = 0.0;

int main(int argc, char* argv[]) {
    double *a, *b, *c; 
    size_t n = atoi(argv[1]); 
    size_t size = n * n * sizeof(double);

    hipMallocManaged((void**)&a, size); 
    hipMallocManaged((void**)&b, size);
    hipMallocManaged((void**)&c, size);

    // initialize host matrix a & b
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            *(a + i * n + j) = i + j;
            *(b + i * n + j) = i + j;
        }
    }

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, a, n, b, n, &beta, c, n);

    hipDeviceSynchronize();
    
    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipblasDestroy(handle);

    return 0;
}
